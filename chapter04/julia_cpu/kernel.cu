// julia_cpu

#include <stdio.h>
#include "../../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
    float r;  // ������ʵ������
    float i;  // ��������������

    hipComplex(float a, float b) :r(a), i(b) { }

    float magnitude2(void)
    {
        return r * r + i * i;  // ������ģ��ƽ��
    }

    hipComplex operator * (const hipComplex& a)
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    hipComplex operator + (const hipComplex& a)
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

int julia(int x, int y)
{
    const float scale = 1.5;
    // DIM / 2 - x��DIM / 2 - y��ԭ�㶨λ��ͼ������
    // ����(DIM / 2)��Ϊ��ȷ��ͼ��ķ�ΧΪ[-1.0, 1.0]
    // scale����������ͼ��ģ����������޸�
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);  // ����C = -0.8 + 0.156i
    hipComplex z(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++)
    {
        z = z * z + c; //Zn+1 = Zn^2 + C
        if (z.magnitude2() > 1000)
        {
            // ����200�Σ�ÿ�ε����궼�жϽ���Ƿ񳬹���ֵ(������1000)����������Ͳ�����julia��
            return 0;
        }
    }
    return 1;  // ����Julia��
}

void kernel(unsigned char* ptr)
{
    for (int y = 0; y < DIM; y++)
    {
        for (int x = 0; x < DIM; x++)
        {
            int offset = x + y * DIM;  // �������ڴ��е�����ƫ�ƣ���Ϊͼ�����ڴ���ʵ����һά�洢��

            int juliaValue = julia(x, y);  // �жϵ�(x, y)�Ƿ�����Julia���ϣ����ڷ���1�������ڷ���0
            // juliaValueΪ0ʱΪ��ɫ(0,0,0)��Ϊ1ʱΪ��ɫ(255,0,0)
            ptr[offset * 4 + 0] = 255 * juliaValue; // redͨ��
            ptr[offset * 4 + 1] = 0;                // greenͨ��
            ptr[offset * 4 + 2] = 0;                // blueͨ��
            ptr[offset * 4 + 3] = 255;              // alphaͨ��
        }
    }
}

int main()
{
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* ptr = bitmap.get_ptr();
    kernel(ptr);  // ��ָ��ͼ���ָ�봫�ݸ��˺���
    bitmap.display_and_exit();

    getchar();
    return 0;
}