#include "hip/hip_runtime.h"
// julia_gpu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
    float   r;
    float   i;
    // hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ hipComplex(float a, float b) : r(a), i(b) { }  // __device__��ʾ���뽫���豸���������������С�
                                                             // ��������Ϊ__device__���������ֻ�ܴ�����__device__
                                                             // ��__global__�����е�������
    __device__ float magnitude2(void)						 
    {														 
        return r * r + i * i;
    }
    __device__ hipComplex operator * (const hipComplex& a)
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator + (const hipComplex& a)
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr) {
    // ��threadIdx/blockIdxӳ�䵽����λ��
    int x = blockIdx.x;
    int y = blockIdx.y;

    int offset = x + y * gridDim.x;  // �����е��߳̿���ԣ�gridDim�ǳ��������������̸߳�ÿһά�Ĵ�С��
                                     // gridDim.x���̸߳�Ŀ��

    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

int main() {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_ptr;

    HANDLE_ERROR(hipMalloc((void**)&dev_ptr, bitmap.image_size()));

    dim3 grid(DIM, DIM);  // ��ά�̸߳�(Grid)��dim3��ʾһ����ά���飬���������ά��ʵ����1��ͬdim3 grid(DIM, DIM, 1)
    kernel<<<grid, 1>>>(dev_ptr);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_ptr, bitmap.image_size(), hipMemcpyDeviceToHost));

    bitmap.display_and_exit();

    HANDLE_ERROR(hipFree(dev_ptr));

    return 0;
}