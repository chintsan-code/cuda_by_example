#include "hip/hip_runtime.h"
// add_loop_gpu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

#define N 10

__global__ void add(int* a, int* b, int* c) {
    int tid = blockIdx.x;  // ��ǰִ�����豸������߳̿�(Block)������,��һ���߳̿������Ϊ0
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];
    int* dev_a, * dev_b, * dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    // <<<b, t>>> b:�豸��ִ�к˺���ʱʹ�õĲ����߳̿�(Block)������t:CUDA Runtime��ÿ���߳̿��д������߳�����
    // N���߳̿� * 1���߳�/�߳̿� = N�������߳�
    // �������˺���ʱ�����ǽ������߳̿�(Block)������ָ��ΪN����������߳̿鼯��Ҳ��Ϊһ���̸߳�(Grid),
    // ���Ǹ���CUDA Runtime��������Ҫһ��һά���̸߳����а���N���߳̿顣
    add<<<N, 1 >>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    return 0;
}