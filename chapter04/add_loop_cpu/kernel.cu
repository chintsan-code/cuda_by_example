//add_loop_cpu


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

void add(int* a, int* b, int* c)
{
    int tid = 0;  // ���ǵ�0��CPU�����������0��ʼ
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += 1;  // ��������ֻ��һ��CPU�����ÿ�ε���1
    }
}

int main()
{
    int a[N], b[N], c[N];

    // ��ʼ����������
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }
    add(a, b, c);
    // ��ӡ���
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}