#include "hip/hip_runtime.h"
// heat_notexture

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"
#include "../../common/cpu_anim.h"

#define DIM 1024
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

__global__ void copy_const_kernel(float* iptr, const float* cptr) {
    // ��threadIdx/BlockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    // ���¶Ȳ�Ϊ0ʱ���Ż�ִ�и��ơ�����Ϊ��ά�ַ���Դ��λ����һ�μ���õ����¶�ֵ
    if (cptr[offset] != 0)
        iptr[offset] = cptr[offset];
}

__global__ void blend_kernel(float* outSrc, const float* inSrc) {
    // ��threadIdx/BlockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    // ÿ��Thread���������һ����Ԫ(һ������)����ȡ��Ӧ��Ԫ�������ڵ�Ԫ���¶�ֵ��
    // Ȼ��ִ�и������㣬���õ�����ֵ���µ���Ӧ�ĵ�Ԫ��
    int left = offset - 1;
    int right = offset + 1;
    if (x == 0)
        left++;  // ��Ե������ͬ
    if (x == DIM - 1)
        right--;

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0)
        top += DIM;
    if (y == DIM - 1)
        bottom -= DIM;

    // ���¹�ʽ��T_new = T_old + k * sum(T_neighbor - T_old)
    outSrc[offset] = inSrc[offset] + SPEED * (inSrc[left] + inSrc[right] 
                                            + inSrc[top] + inSrc[bottom]
                                            - inSrc[offset] * 4);
}

// ���º�������Ҫ��ȫ�ֱ���
struct DataBlock{
    unsigned char* dev_bitmap;
    float* dev_inSrc;  // ���뻺����
    float* dev_outSrc;  // ���������
    float* dev_constSrc;  // ��ʼ������Դ
    CPUAnimBitmap* bitmap;

    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

// ÿһ֡����������anim_gpu()
void anim_gpu(DataBlock* data, int ticks) {
    HANDLE_ERROR(hipEventRecord(data->start, 0));
    
    // ÿ��Block��(16, 16)��Thread��(DIM/16, DIM/16)��֯��һ��Grid
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap* bitmap = data->bitmap;
    
    // ÿһ֡������������90�ֵ������㣬�����޸����ֵ
    for (int i = 0; i < 90; i++) {
        // Ϊ�˼򵥣���Դ��Ԫ������¶Ƚ����ֲ��䡣���ǣ��������ԴӸ��ȵĵ�Ԫ����������ĵ�Ԫ
        copy_const_kernel<<<blocks, threads>>>(data->dev_inSrc, data->dev_constSrc);
        // ����ÿһ����Ԫ
        blend_kernel<<<blocks, threads>>>(data->dev_outSrc, data->dev_inSrc);
        // ������������룬�����μ���������Ϊ�´μ��������
        swap(data->dev_inSrc, data->dev_outSrc);
    }

    // ���¶�תΪ��ɫ
    float_to_color<<<blocks, threads>>>(data->dev_bitmap, data->dev_inSrc);
    // ��������ƻ�CPU
    HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(),
                            data->dev_bitmap,
                            bitmap->image_size(),
                            hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(data->stop, 0));
    HANDLE_ERROR(hipEventSynchronize(data->stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, data->start, data->stop));  // ����ÿһ֡������Ҫ��ʱ��

    data->totalTime += elapsedTime;
    data->frames++;
    printf("Average Time per frame:  %3.1f ms\n", data->totalTime / data->frames);
}

void anim_exit(DataBlock* data) {
    HANDLE_ERROR(hipFree(data->dev_inSrc));
    HANDLE_ERROR(hipFree(data->dev_outSrc));
    HANDLE_ERROR(hipFree(data->dev_constSrc));

    HANDLE_ERROR(hipEventDestroy(data->start));
    HANDLE_ERROR(hipEventDestroy(data->stop));
}

int main() {
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;

    HANDLE_ERROR(hipEventCreate(&data.start));
    HANDLE_ERROR(hipEventCreate(&data.stop));

    HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

    // ����float���͵Ĵ�СΪ4���ַ�(��rgba)
    HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));

    float* temp = (float*)malloc(bitmap.image_size());
    // �������һЩ��Դ
    for (int i = 0; i < DIM*DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
        for (int x = 400; x < 500; x++) {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_constSrc,
                            temp,
                            bitmap.image_size(),
                            hipMemcpyHostToDevice));

    for (int y = 800; y < DIM; y++) {
        for (int x = 0; x < 200; x++) {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_inSrc, 
                            temp,
                            bitmap.image_size(),
                            hipMemcpyHostToDevice));
    free(temp);
    // ÿ����Ҫ����һ֡ͼ�񣬾͵���һ��anim_gpu��֮���ٵ���anim_exit��������Դ��ͷŵ�
    bitmap.anim_and_exit((void (*)(void*, int))anim_gpu,
        (void (*)(void*))anim_exit);

    return 0;
}