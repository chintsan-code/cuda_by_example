#include "hip/hip_runtime.h"
// heat

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"
#include "../../common/cpu_anim.h"

#define DIM 1024
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// �����������ã���Щ������λ��GPU��
texture<float>  texConstSrc;
texture<float>  texIn;
texture<float>  texOut;

__global__ void copy_const_kernel(float* iptr) {
    // ��threadIdx/BlockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    // ���¶Ȳ�Ϊ0ʱ���Ż�ִ�и��ơ�����Ϊ��ά�ַ���Դ��λ����һ�μ���õ����¶�ֵ
    float center = tex1Dfetch(texConstSrc, offset);
    if (center != 0)
        iptr[offset] = center;
}

__global__ void blend_kernel(float* dst, bool dstOut) {
    // ��threadIdx/BlockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    // ÿ��Thread���������һ����Ԫ(һ������)����ȡ��Ӧ��Ԫ�������ڵ�Ԫ���¶�ֵ��
    // Ȼ��ִ�и������㣬���õ�����ֵ���µ���Ӧ�ĵ�Ԫ��
    int left = offset - 1;
    int right = offset + 1;
    if (x == 0)
        left++;  // ��Ե������ͬ
    if (x == DIM - 1)
        right--;

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0)
        top += DIM;
    if (y == DIM - 1)
        bottom -= DIM;

    float t, l, c, r, b;
    if (dstOut) {
        t = tex1Dfetch(texIn, top);     // top
        l = tex1Dfetch(texIn, left);    // left
        c = tex1Dfetch(texIn, offset);  // center
        r = tex1Dfetch(texIn, right);   // right
        b = tex1Dfetch(texIn, bottom);  // bottom
    }
    else {
        t = tex1Dfetch(texOut, top);
        l = tex1Dfetch(texOut, left);
        c = tex1Dfetch(texOut, offset);
        r = tex1Dfetch(texOut, right);
        b = tex1Dfetch(texOut, bottom);
    }
    // ���¹�ʽ��T_new = T_old + k * sum(T_neighbor - T_old)
    dst[offset] = c + SPEED * (t + b + l + r - 4 * c);
}

// ���º�������Ҫ��ȫ�ֱ���
struct DataBlock
{
    unsigned char* dev_bitmap;
    float* dev_inSrc;  // ���뻺����
    float* dev_outSrc;  // ���������
    float* dev_constSrc;  // ��ʼ������Դ
    CPUAnimBitmap* bitmap;

    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

// ÿһ֡����������anim_gpu()
void anim_gpu(DataBlock* data, int ticks) {
    HANDLE_ERROR(hipEventRecord(data->start, 0));

    // ÿ��Block��(16, 16)��Thread��(DIM/16, DIM/16)��֯��һ��Grid
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap* bitmap = data->bitmap;

    // ÿһ֡������������90�ֵ������㣬�����޸����ֵ
    // ����tex��ȫ�ֲ������н�ģ������Ҫͨ��һ����ʶ��ѡ��
    // ÿ�ε������ĸ�������/���
    volatile bool dstOut = true;
    for (int i = 0; i < 90; i++) {
        float *in, *out;
        if (dstOut) {
            in = data->dev_inSrc;
            out = data->dev_outSrc;
        }
        else {
            in = data->dev_outSrc;
            out = data->dev_inSrc;
        }

        // Ϊ�˼򵥣���Դ��Ԫ������¶Ƚ����ֲ��䡣���ǣ��������ԴӸ��ȵĵ�Ԫ����������ĵ�Ԫ
        copy_const_kernel<<<blocks, threads>>>(in);
        // ����ÿһ����Ԫ
        blend_kernel<<<blocks, threads>>>(out, dstOut);
        // ������������룬�����μ���������Ϊ�´μ��������
        dstOut = !dstOut;
    }

    // ���¶�תΪ��ɫ
    float_to_color<<<blocks, threads>>>(data->dev_bitmap, data->dev_inSrc);
    // ��������ƻ�CPU
    HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(),
        data->dev_bitmap,
        bitmap->image_size(),
        hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(data->stop, 0));
    HANDLE_ERROR(hipEventSynchronize(data->stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, data->start, data->stop));  // ����ÿһ֡������Ҫ��ʱ��

    data->totalTime += elapsedTime;
    data->frames++;
    printf("Average Time per frame:  %3.1f ms\n", data->totalTime / data->frames);
}

void anim_exit(DataBlock* data) {
    // ȡ�������ڴ�İ�
    HANDLE_ERROR(hipUnbindTexture(texConstSrc));
    HANDLE_ERROR(hipUnbindTexture(texIn));
    HANDLE_ERROR(hipUnbindTexture(texOut));

    HANDLE_ERROR(hipFree(data->dev_inSrc));
    HANDLE_ERROR(hipFree(data->dev_outSrc));
    HANDLE_ERROR(hipFree(data->dev_constSrc));

    HANDLE_ERROR(hipEventDestroy(data->start));
    HANDLE_ERROR(hipEventDestroy(data->stop));
}

int main() {
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;

    HANDLE_ERROR(hipEventCreate(&data.start));
    HANDLE_ERROR(hipEventCreate(&data.stop));

    HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

    // ����float���͵Ĵ�СΪ4���ַ�(��rgba)
    HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));
    
    // �������ڴ�󶨵�֮ǰ����������Ӧ��
    HANDLE_ERROR(hipBindTexture(NULL, texConstSrc, data.dev_constSrc, bitmap.image_size()));
    HANDLE_ERROR(hipBindTexture(NULL, texIn, data.dev_inSrc, bitmap.image_size()));
    HANDLE_ERROR(hipBindTexture(NULL, texOut, data.dev_outSrc, bitmap.image_size()));

    float* temp = (float*)malloc(bitmap.image_size());
    // �������һЩ��Դ
    for (int i = 0; i < DIM * DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
        for (int x = 400; x < 500; x++) {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_constSrc,
        temp,
        bitmap.image_size(),
        hipMemcpyHostToDevice));

    for (int y = 800; y < DIM; y++) {
        for (int x = 0; x < 200; x++) {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_inSrc,
        temp,
        bitmap.image_size(),
        hipMemcpyHostToDevice));
    free(temp);
    // ÿ����Ҫ����һ֡ͼ�񣬾͵���һ��anim_gpu��֮���ٵ���anim_exit��������Դ��ͷŵ�
    bitmap.anim_and_exit((void (*)(void*, int))anim_gpu,
        (void (*)(void*))anim_exit);

    return 0;
}