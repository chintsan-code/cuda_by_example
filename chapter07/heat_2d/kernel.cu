#include "hip/hip_runtime.h"
// heat_2d

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"
#include "../../common/cpu_anim.h"

#define DIM 1024
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// �����������ã���Щ������λ��GPU��
texture<float, 2>  texConstSrc;
texture<float, 2>  texIn;
texture<float, 2>  texOut;

__global__ void copy_const_kernel(float* iptr) {
    // ��threadIdx/BlockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    // ���¶Ȳ�Ϊ0ʱ���Ż�ִ�и��ơ�����Ϊ��ά�ַ���Դ��λ����һ�μ���õ����¶�ֵ
    float center = tex2D(texConstSrc, x, y);
    if (center != 0)
        iptr[offset] = center;
}

__global__ void blend_kernel(float* dst, bool dstOut) {
    // ��threadIdx/BlockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    float t, l, c, r, b;
    if (dstOut) {
        t = tex2D(texIn, x, y - 1); // top
        l = tex2D(texIn, x - 1, y); // left
        c = tex2D(texIn, x, y);     // center
        r = tex2D(texIn, x + 1, y); // right
        b = tex2D(texIn, x, y + 1); // bottom
    }
    else {
        t = tex2D(texOut, x, y - 1);
        l = tex2D(texOut, x - 1, y);
        c = tex2D(texOut, x, y);
        r = tex2D(texOut, x + 1, y);
        b = tex2D(texOut, x, y + 1);
    }
    // ���¹�ʽ��T_new = T_old + k * sum(T_neighbor - T_old)
    dst[offset] = c + SPEED * (t + b + l + r - 4 * c);
}

// ���º�������Ҫ��ȫ�ֱ���
struct DataBlock
{
    unsigned char* dev_bitmap;
    float* dev_inSrc;  // ���뻺����
    float* dev_outSrc;  // ���������
    float* dev_constSrc;  // ��ʼ������Դ
    CPUAnimBitmap* bitmap;

    hipEvent_t start, stop;
    float totalTime;
    float frames;
};

// ÿһ֡����������anim_gpu()
void anim_gpu(DataBlock* data, int ticks) {
    HANDLE_ERROR(hipEventRecord(data->start, 0));

    // ÿ��Block��(16, 16)��Thread��(DIM/16, DIM/16)��֯��һ��Grid
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    CPUAnimBitmap* bitmap = data->bitmap;

    // ÿһ֡������������90�ֵ������㣬�����޸����ֵ
    // ����tex��ȫ�ֲ������н�ģ������Ҫͨ��һ����ʶ��ѡ��
    // ÿ�ε������ĸ�������/���
    volatile bool dstOut = true;
    for (int i = 0; i < 90; i++) {
        float* in, * out;
        if (dstOut) {
            in = data->dev_inSrc;
            out = data->dev_outSrc;
        }
        else {
            in = data->dev_outSrc;
            out = data->dev_inSrc;
        }

        // Ϊ�˼򵥣���Դ��Ԫ������¶Ƚ����ֲ��䡣���ǣ��������ԴӸ��ȵĵ�Ԫ����������ĵ�Ԫ
        copy_const_kernel<<<blocks, threads>>>(in);
        // ����ÿһ����Ԫ
        blend_kernel<<<blocks, threads>>>(out, dstOut);
        // ������������룬�����μ���������Ϊ�´μ��������
        dstOut = !dstOut;
    }

    // ���¶�תΪ��ɫ
    float_to_color<<<blocks, threads>>>(data->dev_bitmap, data->dev_inSrc);
    // ��������ƻ�CPU
    HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(),
        data->dev_bitmap,
        bitmap->image_size(),
        hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(data->stop, 0));
    HANDLE_ERROR(hipEventSynchronize(data->stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, data->start, data->stop));  // ����ÿһ֡������Ҫ��ʱ��

    data->totalTime += elapsedTime;
    data->frames++;
    printf("Average Time per frame:  %3.1f ms\n", data->totalTime / data->frames);
}

void anim_exit(DataBlock* data) {
    // ȡ�������ڴ�İ�
    HANDLE_ERROR(hipUnbindTexture(texConstSrc));
    HANDLE_ERROR(hipUnbindTexture(texIn));
    HANDLE_ERROR(hipUnbindTexture(texOut));

    HANDLE_ERROR(hipFree(data->dev_inSrc));
    HANDLE_ERROR(hipFree(data->dev_outSrc));
    HANDLE_ERROR(hipFree(data->dev_constSrc));

    HANDLE_ERROR(hipEventDestroy(data->start));
    HANDLE_ERROR(hipEventDestroy(data->stop));
}

int main() {
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;

    HANDLE_ERROR(hipEventCreate(&data.start));
    HANDLE_ERROR(hipEventCreate(&data.stop));

    HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

    // ����float���͵Ĵ�СΪ4���ַ�(��rgba)
    HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
    HANDLE_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));

    // �������ڴ�󶨵�֮ǰ����������Ӧ��
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    HANDLE_ERROR(hipBindTexture2D(NULL, texConstSrc, data.dev_constSrc, desc, DIM, DIM, sizeof(float) * DIM));
    HANDLE_ERROR(hipBindTexture2D(NULL, texIn, data.dev_inSrc, desc, DIM, DIM, sizeof(float) * DIM));
    HANDLE_ERROR(hipBindTexture2D(NULL, texOut, data.dev_outSrc, desc, DIM, DIM, sizeof(float) * DIM));

    float* temp = (float*)malloc(bitmap.image_size());
    // �������һЩ��Դ
    for (int i = 0; i < DIM * DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
        for (int x = 400; x < 500; x++) {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_constSrc,
        temp,
        bitmap.image_size(),
        hipMemcpyHostToDevice));

    for (int y = 800; y < DIM; y++) {
        for (int x = 0; x < 200; x++) {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR(hipMemcpy(data.dev_inSrc,
        temp,
        bitmap.image_size(),
        hipMemcpyHostToDevice));
    free(temp);
    // ÿ����Ҫ����һ֡ͼ�񣬾͵���һ��anim_gpu��֮���ٵ���anim_exit��������Դ��ͷŵ�
    bitmap.anim_and_exit((void (*)(void*, int))anim_gpu,
        (void (*)(void*))anim_exit);

    return 0;
}