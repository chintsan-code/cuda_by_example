#include "hip/hip_runtime.h"
// dot

#ifdef __INTELLISENSE__

// in here put whatever is your favorite flavor of intellisense workarounds
void __syncthreads(void);

#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

#define imin(a,b) (a<b?a:b)
#define sum_squares(x)  (x*(x+1)*(2*x+1)/6)

// Ĭ������32��Block��ÿ��Block����256��Thread
// ��NС��8192ʱ�����������ͻ����Thread���˷ѣ���Ҫ����������Block������
// ��NС�ڻ����8192ʱ��ÿ��Thread����һ��ʸ��Ԫ�أ���N����8192ʱ������Thread������ʸ��Ԫ��
const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float* a, float* b, float* c) {
    __shared__ float cache[threadsPerBlock];  // ʹ�ùؼ���__shared__����һ������פ���ڹ����ڴ���
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        // һ��Thread��Ҫ������ʸ��Ԫ�صĳ˻�
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    // cache����ӦthreadIdx��Thread�����Ķ��ʸ��Ԫ�صĳ˻��ۼ�����
    cache[cacheIndex] = temp;

    // ��Block�е�Thread����ͬ��
    __syncthreads();

    // ��Լ(Reduction)�㷨���
    // ���ڹ�Լ������˵�����´���Ҫ��threadsPerBlock������2�ı���
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main() {
    float *a, *b, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float c;

    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float)));

    // ������ʸ����ʼ��Ϊ��������У�����֮��ʹ��ƽ����������͹�ʽ��֤
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

    // ��CPU��������յ��������
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++)
    {
        c += partial_c[i];  // partial_c[i]���ǵ�i��Block���ص�Ԫ�س˻�֮�ͣ�һ����blocksPerGrid��Block�������Ǽ������õ����ս��
    }

    // ��֤�����Ӧ���빫ʽ����Ľ��һ��
    printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares((float)(N - 1)));

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_partial_c));

    free(a);
    free(b);
    free(partial_c);

    return 0;
}