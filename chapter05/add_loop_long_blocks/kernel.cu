#include "hip/hip_runtime.h"
// add_loop_long_blocks

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

#define N (33*1024)  // ��ô���N�����ֻ��һ��Block�������˵�

__global__ void add(int* a, int* b, int* c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

// ��N�ܴ�ʱҪ�޸���Ŀ����->��������->������->ϵͳ->��ջ������С(��λ: byte)
// ���߷�����ȫ���������
// int a[N], b[N], c[N];  // ������ȫ����
int main() {
    //int a[N], b[N], c[N];  // ������ջ��
    int* a, * b, * c;
    a = (int*)malloc(N * sizeof(int));  // �����ڶ��������ǵ�Ҫ�ͷ�
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));

    int* dev_a, * dev_b, * dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    // (127 + N) / 128 : N/128����ȡ��
    // ÿ��Block��128��Thread
    // ע��( N + 127 ) / 128���ܳ���maxGridSize������
    add<<<(127 + N) / 128, 128>>>(dev_a, dev_b, dev_c);  

    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    free(a);  // �ͷ�֮ǰ����Ķ����ڴ�
    free(b);
    free(c);

    return 0;
}