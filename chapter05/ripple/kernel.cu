#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"
#include "../../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel(unsigned char* ptr, int ticks) {
    // ��threadIdx/blockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int offset = y * blockDim.x * gridDim.x + x;

    // ����Ĵ����붯���йأ����ù���
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                                         cos(d / 10.0f - ticks / 7.0f) /
                                         (d / 10.0f + 1.0f));
    ptr[offset * 4 + 0] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;

}

struct DataBlock
{
    unsigned char* dev_bitmap;
    CPUAnimBitmap* bitmap;
};

void generate_frame(DataBlock* d, int ticks) {
    // (DIM/16, DIM/16)��Block���һ��Grid
    // ÿ��Block����(16, 16)��Thread
    // ����һ����(DIM, DIM)��Thread����ӦDIM*DIM�ߴ��ͼ��ÿһ��������һ��Thread����
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    kernel<<<blocks, threads>>>(d->dev_bitmap, ticks);

    HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(),
                            d->dev_bitmap,
                            d->bitmap->image_size(),
                            hipMemcpyDeviceToHost));
}

// �ͷ���GPU�Ϸ�����Դ�
void cleanup(DataBlock* d) {
    HANDLE_ERROR(hipFree(d->dev_bitmap));
}

int main() {
    DataBlock data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;

    HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

    // ÿ������һ֡ͼ�񣬵���һ��generate_frame��֮�󽫷�����Դ��ͷŵ�
    bitmap.anim_and_exit((void(*)(void*, int))generate_frame,
                         (void(*)(void*))cleanup);

    return 0;
}