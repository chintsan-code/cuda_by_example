#include "hip/hip_runtime.h"
// add_loop_very_long_blocks

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

#define N (65536*1024)

__global__ void add(int* a, int* b, int* c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid +=  gridDim.x * blockDim.x;  // ������������Thread������ = ������Block����(gridDim.x) * ÿ��Block������Thread������(blockDim.x)
                                         // ��ÿ��Thread���� N / (gridDim.x * blockDim.x)������
    }
}

int main() {
    int* a, * b, * c;
    a = (int*)malloc(N * sizeof(int));
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));

    int* dev_a, * dev_b, * dev_c;
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    // ����128��Block��ÿ��Block����128���߳�
    // ��ÿ��Thread���� N / 16384������
    add<<<128, 128>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    free(a);
    free(b);
    free(c);

    return 0;
}