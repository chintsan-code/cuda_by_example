#include "hip/hip_runtime.h"
// add_loop_blocks

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

#define N 1024

__global__ void add(int* a, int* b, int* c) {
    int tid = threadIdx.x;  // ֻ��һ��Block��ͨ���߳������������ݽ�������
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];
    int* dev_a, * dev_b, * dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<1, N>>>(dev_a, dev_b, dev_c);  // ����1��Block��������N��Thread

    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    return 0;
}