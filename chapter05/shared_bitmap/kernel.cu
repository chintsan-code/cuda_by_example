#include "hip/hip_runtime.h"
// shared_bitmap

#ifdef __INTELLISENSE__

// in here put whatever is your favorite flavor of intellisense workarounds
void __syncthreads(void);

#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "math.h"
#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel(unsigned char* ptr) {
    // ��threadIdx/blockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    // ����һ��������������ΪҪ��CUDA Runtime������block��(16,16)���̣߳����л�������СҲ����Ϊ16*16��
    // ��ÿ���߳��ڸû������ж���һ����Ӧ��λ��
    __shared__ float shared[16][16];

    // ���ڼ������λ���ϵĵ�ֵ
    const float period = 128.0f;
    
    shared[threadIdx.x][threadIdx.y] =
        255 * (sinf(x * 2.0f * PI / period) + 1.0f) *
        (sinf(y * 2.0f * PI / period) + 1.0f) / 4.0f;

    __syncthreads();
    // ��󣬰���Щֵ��������أ�����x��y�Ĵ���
    ptr[offset * 4 + 0] = 0;
    // ע�����������Ϊ(threadIdx.x, threadIdx.y)��Thread��ɶԻ�����shared��д���Ҫ�������
    // shared[15 - threadIdx.x][15 - threadIdx.y]���ж�ȡʱ��
    // ����Ϊ(15 - threadIdx.x, 15 - threadIdx.y)��Thread���ܻ�û��ɶԻ�����shared��д�룬
    // �����Ҫ��֮ǰ����__syncthreads();
    ptr[offset * 4 + 1] = shared[15 - threadIdx.x][15 - threadIdx.y];
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

int main() {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_ptr;

    HANDLE_ERROR(hipMalloc((void**)&dev_ptr, bitmap.image_size()));

    dim3 threads(16, 16);
    dim3 blocks(DIM / 16, DIM / 16);
    kernel<<<blocks, threads>>>(dev_ptr);
    
    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_ptr, bitmap.image_size(), hipMemcpyDeviceToHost));

    bitmap.display_and_exit();

    HANDLE_ERROR(hipFree(dev_ptr));

    return 0;
}