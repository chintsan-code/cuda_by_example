#include "hip/hip_runtime.h"
// hist_cpu

#include "../../common/book.h"
#include "time.h"

#define SIZE (100*1024*1024)

int main() {
    // �������100MB���������
    unsigned char* buffer = (unsigned char*)big_random_block(SIZE);
    // ÿ���ֽڵ�ȡֵ��ΧΪ0x00-0xFF,���ʹ�ô�СΪ256���������洢��Ӧ��ֵ��buffer�г��ֵĴ���,
    // ���ڼ���ֱ��ͼ
    unsigned int histo[256] = { 0 };
    // ����ʱ��
    clock_t start, stop;
    start = clock();
    for (int i = 0; i < SIZE; i++) {
        histo[buffer[i]]++;
    }
    stop = clock();
    float elapsedTime = (float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f;
    printf("Time to generate:  %3.1f ms\n", elapsedTime);
    // ��ֱ֤��ͼ������Ԫ�ؼ������Ƿ������ȷ��ֵ(Ӧ�õ���SIZE)
    long histoCount = 0;
    for (int i = 0; i < 256; i++) {
        histoCount += histo[i];
    }
    printf("Histogram Sum:  %ld\n", histoCount);

    // �ͷ��ڴ�
    free(buffer);

    return 0;
}