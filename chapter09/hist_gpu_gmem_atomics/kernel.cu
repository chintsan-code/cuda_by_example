#include "hip/hip_runtime.h"
// hist_gpu_gmem_atomics

#include "hip/hip_runtime.h"
#include ""

#include "../../common/book.h"

#include <stdio.h>

#define SIZE (100*1024*1024)

__global__ void histo_kernel(unsigned char* buffer,
                             long size, 
                             unsigned int* histo) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    while (tid < size) {
        // ��ʾ��CUDA C��ʹ��ԭ�Ӳ����ķ�ʽ����������atomicAdd(address, val)
        // ������һ��ԭ�ӵ����в���������������а�����ȡ��ַaddress����ֵ����val���ӵ����ֵ�ϣ�
        // �Լ����������ص�ַaddress���ײ�Ӳ����ȷ����ִ����Щ����ʱ��
        // �����κ��̶߳������ȡ��д���ַaddress�ϵ�ֵ����������ȷ���õ�Ԥ�ƵĽ����
        atomicAdd(&(histo[buffer[tid]]), 1);
        tid += stride;
    }
}

int main() {
    // �������100MB���������
    unsigned char* buffer = (unsigned char*)big_random_block(SIZE);

    // ��ʼ����ʱ�¼�
    hipEvent_t start, end;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // ��GPU��Ϊ�ļ������ݷ����ڴ�
    unsigned char* dev_buffer;
    unsigned int* dev_histo;
    HANDLE_ERROR(hipMalloc((void**)&dev_buffer, SIZE * sizeof(char)));
    HANDLE_ERROR(hipMalloc((void**)&dev_histo, 256 * sizeof(int)));
    HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE * sizeof(char), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemset(dev_histo, 0, 256 * sizeof(int)));

    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount * 2; // ��Block����������ΪGPU�д�����������2��
    histo_kernel<<<blocks, 256>>>(dev_buffer, SIZE, dev_histo);

    unsigned int histo[256];
    HANDLE_ERROR(hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost));

    // �õ�ֹͣʱ�䲢��ʾ��ʱ���
    HANDLE_ERROR(hipEventRecord(end, 0));
    HANDLE_ERROR(hipEventSynchronize(end));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, end));
    printf("Time to generate:  %3.1f ms\n", elapsedTime);

    // ��ֱ֤��ͼ������Ԫ�ؼ������Ƿ������ȷ��ֵ(Ӧ�õ���SIZE)
    long histoCount = 0;
    for (int i = 0; i < 256; i++) {
        histoCount += histo[i];
    }
    printf("Histogram Sum:  %ld\n", histoCount);

    // ��֤��CPU�õ�������ͬ�ļ���ֵ
    for (int i = 0; i < SIZE; i++) {
        histo[buffer[i]]--;
    }
    for (int i = 0; i < 256; i++) {
        if (histo[i] != 0) {
            printf("Failure at %d!  Off by %d\n", i, histo[i]);
        }
    }

    // �ͷ��¼����ڴ�
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));
    HANDLE_ERROR(hipFree(dev_buffer));
    HANDLE_ERROR(hipFree(dev_histo));
    free(buffer);

    return 0;
}