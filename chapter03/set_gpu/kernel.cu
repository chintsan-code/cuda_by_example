// set_gpu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

int main()
{
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of current CUDA device:  %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;//����ѡ������������>1.3
    prop.minor = 3;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));//������ƥ����豸id(�������豸��û�ﵽ������Ҳ�᷵��һ����ƥ���)
    printf("ID of CUDA device closest to revision 1.3:  %d\n", dev);

    HANDLE_ERROR(hipSetDevice(dev));//����GPU�豸��֮�����е��豸���������ڴ��豸��ִ��

    return 0;
}
