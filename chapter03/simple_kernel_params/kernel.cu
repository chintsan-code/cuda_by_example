#include "hip/hip_runtime.h"
// simple_kernel_params

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

int main() {
	int c;
	int* dev_c;
	// �����Դ�
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
	// �����C�����еĺ���һ�����ú˺���
	add <<<1, 1 >>> (2, 7, dev_c);
	// ��������ֱ�Ӷ�dev_c��ָ���Դ���������Ӧ�ø��ƻ������ڴ�
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

	printf("2 + 7 = %d\n", c);
	// ���Ҫ�ͷ�֮ǰ������Դ�
	hipFree(dev_c);

	return 0;
}