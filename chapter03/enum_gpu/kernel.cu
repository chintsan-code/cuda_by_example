// enum_gpu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../../common/book.h"

int main() {
    hipDeviceProp_t prop;

    int count;
    // ��ȡCUDA�豸������
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        // �豸���
        printf("   --- General Information for device %d ---\n", i);
        // ��ʶ�豸��ASCII�ַ���
        printf("Name:  %s\n", prop.name);
        // �豸������
        printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
        // ʱ��Ƶ��(��λ: kHz)
        printf("Clock rate:  %d\n", prop.clockRate);
        // �豸�Ƿ����ͬʱ�����ڴ沢ִ���ں�
        printf("Device copy overlap:  ");
        if (prop.deviceOverlap)
            printf("Enabled\n");
        else
            printf("Disabled\n");
        // ָ���ں��Ƿ�������ʱ������
        printf("Kernel execution timeout :  ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled\n");
        else
            printf("Disabled\n");

        printf("   --- Memory Information for device %d ---\n", i);
        // �豸�Ͽ��õ�ȫ���ڴ�(��λ: byte)
        printf("Total global mem:  %ld\n", prop.totalGlobalMem);
        // �豸�Ͽ��õĺ㶨�ڴ�(��λ: byte)
        printf("Total constant Mem:  %ld\n", prop.totalConstMem);
        // ���ڴ渴�Ƶ�����������(��λ: byte)
        printf("Max mem pitch:  %ld\n", prop.memPitch);
        // ����Ķ���Ҫ��
        printf("Texture Alignment:  %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d ---\n", i);
        // �豸�ϵĶദ��������
        printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
        // ÿ���߳̿�(Block)���õĹ����ڴ�(��λ: byte)
        printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
        // ÿ���߳̿�(Block)����32λ�Ĵ���
        printf("Registers per mp:  %d\n", prop.regsPerBlock);
        // ��һ���߳���(Warp)�а������߳�����
        printf("Threads in warp:  %d\n", prop.warpSize);
        // ÿһ���߳̿�(Block)�ɰ���������߳�����
        printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
        // �ڶ�ά�߳̿�(Block)�����У�ÿһά���԰������߳̿�����
        printf("Max thread dimensions:  (%d, %d, %d)\n",
            prop.maxThreadsDim[0],
            prop.maxThreadsDim[1],
            prop.maxThreadsDim[2]);
        // ��ÿһ���̸߳�(Grid)�У�ÿһά���԰������߳̿�(Block)����
        printf("Max grid dimensions:  (%d, %d, %d)\n",
            prop.maxGridSize[0],
            prop.maxGridSize[1],
            prop.maxGridSize[2]);

        printf("\n");
    }

    return 0;
}
