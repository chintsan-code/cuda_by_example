#include "hip/hip_runtime.h"
// basic_single_stream

#include "hip/hip_runtime.h"
#include ""

#include "../../common/book.h"
#include <stdio.h>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int* a, int* b, int* c) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        int id1 = (id + 1) % 256;
        int id2 = (id + 2) % 256;
        float as = (a[id] + a[id1] + a[id2]) / 3.0f;
        float bs = (b[id] + b[id1] + b[id2]) / 3.0f;
        c[id] = (as + bs) / 2;
    }
}

int main() {
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
    // ѡ��һ��֧���豸�ص�(Device Overlap)���ܵ��豸:
    // �ܹ���ִ��һ��CUDA C�˺�����ͬʱ���������豸������֮��ִ�и��Ʋ���
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no "
            "speed up from stream\n"); 
    }

    hipEvent_t start, end;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // ��ʼ��Stream
    hipStream_t stream;
    HANDLE_ERROR(hipStreamCreate(&stream));

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // ��GPU�Ϸ����ڴ�
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // ������Streamʹ�õ�Page-Locked�ڴ�
    HANDLE_ERROR(hipHostAlloc((void**)&host_a,
                                FULL_DATA_SIZE * sizeof(int),
                                hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b,
                                FULL_DATA_SIZE * sizeof(int),
                                hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c,
                                FULL_DATA_SIZE * sizeof(int),
                                hipHostMallocDefault));

    // ʹ�����������������ڴ�
    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // ������������ѭ����ÿ�����ݿ�Ĵ�СΪN
    for (int offset = 0; offset < FULL_DATA_SIZE; offset +=N) {
        // ��Page-Locked Memory���첽��ʽ���Ƶ�Device��
        // ��һ�θ���
        HANDLE_ERROR(hipMemcpyAsync(dev_a, 
                                host_a + offset,			// ����һ��ƫ��offset
                                N * sizeof(int), 
                                hipMemcpyHostToDevice,
                                stream));					// �����stream�н��и���
        // �ڶ��θ���
        HANDLE_ERROR(hipMemcpyAsync(dev_b,
                                host_b + offset, 
                                N * sizeof(int),
                                hipMemcpyHostToDevice,
                                stream));

        kernel<<<N / 256, 256>>>(dev_a, dev_b, dev_c);
        
        // �����θ���
        // �����ݴ�Device���Ƶ�Page-Locked Memory
        HANDLE_ERROR(hipMemcpyAsync(host_c + offset,
                                dev_c,
                                N * sizeof(int),
                                hipMemcpyDeviceToHost,
                                stream));
    }

    // ����������ҳ�����ڴ渴�Ƶ������ڴ�
    HANDLE_ERROR(hipStreamSynchronize(stream));

    HANDLE_ERROR(hipEventRecord(end, 0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, end));

    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));
    HANDLE_ERROR(hipStreamDestroy(stream));  // �ͷ���

    return 0;
}