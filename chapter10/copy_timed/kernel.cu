// copy_timed

#include "hip/hip_runtime.h"
#include ""

#include "../../common/book.h"

#include <stdio.h>

#define SIZE    (64*1024*1024)

float cuda_malloc_test(int size, bool up) {
    hipEvent_t start, end;
    int *a, *dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    
    // ����������������GPU������
    a = (int*)malloc(size * sizeof(int));  // ʹ�ñ�׼C����malloc()������ɷ�ҳ�����ڴ�
    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(int)));

    HANDLE_ERROR(hipEventRecord(start, 0));
    // ִ��100�θ��Ʋ��������ɲ���upָ�����Ʒ���
    for (int i = 0; i < 100; i++) {
        if (up) {
            // hipMemcpyHostToDevice
            HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
        }else{
            // hipMemcpyDeviceToHost
            HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost));
        }
    }
    HANDLE_ERROR(hipEventRecord(end, 0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, end));

    free(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));

    return elapsedTime;
}

float cuda_host_alloc_test(int size, bool up) {
    hipEvent_t start, end;
    int* a, * dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));

    // ����������������GPU������
    HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(int), hipHostMallocDefault));  // ʹ��hipHostAlloc()������̶��ڴ�
    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(int)));

    HANDLE_ERROR(hipEventRecord(start, 0));
    // ִ��100�θ��Ʋ��������ɲ���upָ�����Ʒ���
    for (int i = 0; i < 100; i++) {
        if (up) {
            // hipMemcpyHostToDevice
            HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
        }
        else {
            // hipMemcpyDeviceToHost
            HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost));
        }
    }
    HANDLE_ERROR(hipEventRecord(end, 0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, end));

    HANDLE_ERROR(hipHostFree(a));  // ʹ��hipHostFree()�ͷ���hipHostAlloc()������ڴ�
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));

    return elapsedTime;
}

int main() {
    float elapsedTime;
    float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

    // ���Դ�Host��Device�ĸ�������(ʹ��malloc������ڴ�)
    elapsedTime = cuda_malloc_test(SIZE, true);
    printf("Time using malloc: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy up: %3.1f\n", MB / (elapsedTime / 1000));

    // ���Դ�Device��Host�ĸ�������(ʹ��malloc������ڴ�)
    elapsedTime = cuda_malloc_test(SIZE, false);
    printf("Time using malloc: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy down: %3.1f\n", MB / (elapsedTime / 1000));

    // ���Դ�Host��Device�ĸ�������(ʹ��hipHostAlloc������ڴ�)
    elapsedTime = cuda_host_alloc_test(SIZE, true);
    printf("Time using hipHostAlloc: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy up: %3.1f\n", MB / (elapsedTime / 1000));

    // ���Դ�Device��Host�ĸ�������(ʹ��hipHostAlloc������ڴ�)
    elapsedTime = cuda_host_alloc_test(SIZE, false);
    printf("Time using hipHostAlloc: %3.1f ms\n", elapsedTime);
    printf("\tMB/s during copy down: %3.1f\n", MB / (elapsedTime / 1000));


    return 0;
}