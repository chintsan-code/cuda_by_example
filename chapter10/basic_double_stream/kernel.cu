#include "hip/hip_runtime.h"
// basic_double_stream

#include "hip/hip_runtime.h"
#include ""

#include "../../common/book.h"

#include <stdio.h>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int* a, int* b, int* c) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        int id1 = (id + 1) % 256;
        int id2 = (id + 2) % 256;
        float as = (a[id] + a[id1] + a[id2]) / 3.0f;
        float bs = (b[id] + b[id1] + b[id2]) / 3.0f;
        c[id] = (as + bs) / 2;
    }
}

int main() {
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
    // ѡ��һ��֧���豸�ص�(Device Overlap)���ܵ��豸:
    // �ܹ���ִ��һ��CUDA C�˺�����ͬʱ���������豸������֮��ִ�и��Ʋ���
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no "
            "speed up from stream\n");
    }

    hipEvent_t start, end;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // ��ʼ����
    hipStream_t stream0, stream1;
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));

    int* host_a, * host_b, * host_c;
    int* dev_a0, * dev_b0, * dev_c0;  // Ϊstream0�����GPU�ڴ�
    int* dev_a1, * dev_b1, * dev_c1;  // Ϊstream1�����GPU�ڴ�

    // ��GPU�Ϸ����ڴ�
    HANDLE_ERROR(hipMalloc((void**)&dev_a0, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b0, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c0, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_a1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c1, N * sizeof(int)));

    // ������Streamʹ�õ�Page-Locked�ڴ�
    HANDLE_ERROR(hipHostAlloc((void**)&host_a,
                                FULL_DATA_SIZE * sizeof(int),
                                hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b,
                                FULL_DATA_SIZE * sizeof(int),
                                hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c,
                                FULL_DATA_SIZE * sizeof(int),
                                hipHostMallocDefault));

    // ʹ�����������������ڴ�
    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // ������������ѭ����ÿ�����ݿ�Ĵ�СΪN
    for (int offset = 0; offset < FULL_DATA_SIZE; offset+=N*2) {  // ��2���������ÿ��ѭ����ƫ��2N
        // ��Page-Locked Memory���첽��ʽ���Ƶ�Device��
        // ��һ�θ���(stream0)
        HANDLE_ERROR(hipMemcpyAsync(dev_a0,
                                     host_a + offset,
                                     N * sizeof(int),
                                     hipMemcpyHostToDevice,
                                     stream0));
        // �ڶ��θ���(stream0)
        HANDLE_ERROR(hipMemcpyAsync(dev_b0,
                                     host_b + offset,
                                     N * sizeof(int),
                                     hipMemcpyHostToDevice,
                                     stream0));
        // ִ�к˺���
        kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);

        // �����ݴ�Device���ƻ�Page-Locked Memory
        HANDLE_ERROR(hipMemcpyAsync(host_c + offset,
                                     dev_c0,
                                     N * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     stream0));

        // �����θ���(stream1)
        HANDLE_ERROR(hipMemcpyAsync(dev_a1,
                                     host_a + offset + N,
                                     N * sizeof(int),
                                     hipMemcpyHostToDevice,
                                     stream1));
       
        // ���Ĵθ���(stream1)
        HANDLE_ERROR(hipMemcpyAsync(dev_b1,
                                     host_b + offset + N,
                                     N * sizeof(int),
                                     hipMemcpyHostToDevice,
                                     stream1));

        // ִ�к˺���
        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

        // �����ݴ�Device���ƻ�Page-Locked Memory
        HANDLE_ERROR(hipMemcpyAsync(host_c + offset + N,
                                     dev_c1,
                                     N * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     stream1));
    }
    // ��������ͬ��
    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));

    HANDLE_ERROR(hipEventRecord(end, 0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, end));
    printf("Time taken: %3.1f ms\n", elapsedTime);

    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a0));
    HANDLE_ERROR(hipFree(dev_b0));
    HANDLE_ERROR(hipFree(dev_c0));
    HANDLE_ERROR(hipFree(dev_a1));
    HANDLE_ERROR(hipFree(dev_b1));
    HANDLE_ERROR(hipFree(dev_c1));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));
    HANDLE_ERROR(hipStreamDestroy(stream0));  // �ͷ���
    HANDLE_ERROR(hipStreamDestroy(stream1));

	return 0;
}