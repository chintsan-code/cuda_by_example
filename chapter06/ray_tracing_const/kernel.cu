#include "hip/hip_runtime.h"
// ray_tracing_const

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define INF 2e10f
#define rnd( x ) (x * rand() / RAND_MAX)
#define DIM 1024
#define SPHERES 20

struct Sphere
{
    float r, g, b;  //	�������ɫ
    float radius;   // ����뾶
    float x, y, z;  // �������������(x,y,z)

    // ��������(ox,oy)���Ĺ��ߣ������Ƿ������������ཻ��
    // ����ཻ�����������������������洦�ľ���
    __device__ float hit(float ox, float oy, float* n) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius) {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

// �����ڴ棬��Ҫ��̬����
__constant__ Sphere dev_s[SPHERES];

__global__ void kernel(unsigned char* ptr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;

    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = dev_s[i].hit(ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = dev_s[i].r * fscale;
            g = dev_s[i].g * fscale;
            b = dev_s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}


int main() {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_ptr;


    HANDLE_ERROR(hipMalloc((void**)&dev_ptr, bitmap.image_size()));

    Sphere* spheres = (Sphere*)malloc(SPHERES * sizeof(Sphere));
    for (int i = 0; i < SPHERES; i++) {
        spheres[i].r = rnd(1.0f);
        spheres[i].g = rnd(1.0f);
        spheres[i].b = rnd(1.0f);
        spheres[i].x = rnd(1000.0f) - 500;
        spheres[i].y = rnd(1000.0f) - 500;
        spheres[i].z = rnd(1000.0f) - 500;
        spheres[i].radius = rnd(100.0f) + 20;
    }
    // ʹ��������ڴ濽������hipMemcpyToSymbol()�������ڴ濽����GPU��ʹ�÷�����hipMemcpy����
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_s), spheres, SPHERES * sizeof(Sphere)));
    free(spheres);

    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    kernel<<<blocks, threads>>>(dev_ptr);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_ptr,
        bitmap.image_size(),
        hipMemcpyDeviceToHost));
    bitmap.display_and_exit();

    //HANDLE_ERROR(hipFree(dev_s)); ����Ҫʹ��hipFree()�Գ����ڴ�dev_s�����ͷ�
    HANDLE_ERROR(hipFree(dev_ptr));

    return 0;
}