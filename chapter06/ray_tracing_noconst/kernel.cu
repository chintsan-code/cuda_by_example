#include "hip/hip_runtime.h"
// ray_tracing_noconst

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define INF 2e10f
#define rnd( x ) (x * rand() / RAND_MAX)
#define DIM 1024
#define SPHERES 20

struct Sphere
{
    float r, g, b;  //	�������ɫ
    float radius;   // ����뾶
    float x, y, z;  // �������������(x,y,z)

    // ��������(ox,oy)���Ĺ��ߣ������Ƿ������������ཻ��
    // ����ཻ�����������������������洦�ľ���
    __device__ float hit(float ox, float oy, float* n) {
        float dx = ox - x;
        float dy = oy - y;
        // �ж�ֱ���������ཻ�����
        if (dx * dx + dy * dy < radius * radius) {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__global__ void kernel(Sphere* s, unsigned char* ptr) {
    // ��threadIdx/blockIdxӳ�䵽����λ��
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * gridDim.x * blockDim.x + x;
    // ��ͼ������(x, y)ƫ��DIM/2������z�Ὣ����ͼ�������
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    // ÿ�����߶���Ҫ�ж��������ཻ�������ʹ�õ����ķ�ʽ����hit�����ж�
    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        // ������������˵�ǰ��ǰ�����棬��ô�����ж�����λ�������֮��ľ����Ƿ����һ�����еľ�����ӽ����
        // ������ӽӽ�����ô��������뱣��Ϊ�µ���ӽ����棬������������rgb��ɫֵ��
        // �����ѭ������ʱ����ǰ�߳̾ͻ�֪���������ӽ����������ɫ��
        // ���û�����У���rgbΪ��ʼֵ(0,0,0)
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}


int main() {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_ptr;
    Sphere* dev_s;

    // ��GPU�Ϸ����ڴ��Լ������λͼ
    HANDLE_ERROR(hipMalloc((void**)&dev_ptr, bitmap.image_size()));
    // ΪSphere���ݼ������ڴ�
    HANDLE_ERROR(hipMalloc((void**)&dev_s, SPHERES * sizeof(Sphere)));

    // ������ʱ�ڴ棬�����ʼ���������Ƶ�GPU�ϵ��ڴ棬Ȼ�����ͷ���ʱ�ڴ�
    Sphere* spheres = (Sphere*)malloc(SPHERES * sizeof(Sphere));
    // ����SPHERES���������
    for (int i = 0; i < SPHERES; i++) {
        spheres[i].r = rnd(1.0f);
        spheres[i].g = rnd(1.0f);
        spheres[i].b = rnd(1.0f);
        spheres[i].x = rnd(1000.0f) - 500;
        spheres[i].y = rnd(1000.0f) - 500;
        spheres[i].z = rnd(1000.0f) - 500;
        spheres[i].radius = rnd(100.0f) + 20;
    }
    HANDLE_ERROR(hipMemcpy(dev_s, spheres, SPHERES * sizeof(Sphere), hipMemcpyHostToDevice));
    free(spheres);  // ���Ƶ�GPU��Ϳ����ͷ���ʱ��������

    // ����������������һ��bitmap
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    kernel<<<blocks, threads>>>(dev_s, dev_ptr);

    // ��bitmap��GPU���ƻ�CPU����ʾ
    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_ptr,
        bitmap.image_size(),
        hipMemcpyDeviceToHost));
    bitmap.display_and_exit();

    // �ͷ��ڴ�
    HANDLE_ERROR(hipFree(dev_s));
    HANDLE_ERROR(hipFree(dev_ptr));

    return 0;
}